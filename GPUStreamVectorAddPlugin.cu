 

#include "GPUStreamVectorAddPlugin.h"

void GPUStreamVectorAddPlugin::input(std::string infile) {
  readParameterFile(infile);
}

void GPUStreamVectorAddPlugin::run() {}

void GPUStreamVectorAddPlugin::output(std::string outfile) {
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  hipStream_t stream[4];
  float *d_A[4], *d_B[4], *d_C[4];
  int i, k, Seglen = 1024;
  int Gridlen = (Seglen - 1) / 256 + 1;

 inputLength = atoi(myParameters["N"].c_str());
 hostInput1 = (float*) malloc(inputLength*sizeof(float));
 hostInput2 = (float*) malloc(inputLength*sizeof(float));
 std::ifstream myinput((std::string(PluginManager::prefix())+myParameters["vector1"]).c_str(), std::ios::in);
 for (i = 0; i < inputLength; ++i) {
        float k;
        myinput >> k;
        hostInput1[i] = k;
 }
 std::ifstream myinput2((std::string(PluginManager::prefix())+myParameters["vector2"]).c_str(), std::ios::in);
 for (i = 0; i < inputLength; ++i) {
        float k;
        myinput2 >> k;
        hostInput2[i] = k;
 }


  hostOutput = (float *)malloc((4*Seglen+inputLength) * sizeof(float));
  
  
  for (i = 0; i < 4; i++) {
    hipStreamCreate(&stream[i]);
    hipMalloc((void **)&d_A[i], (4*Seglen+inputLength) * sizeof(float));
    hipMalloc((void **)&d_B[i], (4*Seglen+inputLength) * sizeof(float));
    hipMalloc((void **)&d_C[i], (4*Seglen+inputLength) * sizeof(float));
  }

  for (i = 0; i < inputLength; i += Seglen * 4) {
    for (k = 0; k < 4; k++) {
      hipMemcpyAsync(d_A[k], hostInput1 + i + k * Seglen,
                      Seglen * sizeof(float), hipMemcpyHostToDevice,
                      stream[k]);
      hipMemcpyAsync(d_B[k], hostInput2 + i + k * Seglen,
                      Seglen * sizeof(float), hipMemcpyHostToDevice,
                      stream[k]);
      vecAdd<<<Gridlen, 256, 0, stream[k]>>>(d_A[k], d_B[k], d_C[k],
                                             Seglen);
    }
    hipStreamSynchronize(stream[0]);
    hipStreamSynchronize(stream[1]);
    hipStreamSynchronize(stream[2]);
    hipStreamSynchronize(stream[3]);
    for (k = 0; k < 4; k++) {
      hipMemcpyAsync(hostOutput + i + k * Seglen, d_C[k],
                      Seglen * sizeof(float), hipMemcpyDeviceToHost,
                      stream[k]);
    }
  }
  hipDeviceSynchronize();
 std::ofstream outsfile(outfile.c_str(), std::ios::out);
        for (i = 0; i < inputLength; ++i){
                outsfile << hostOutput[i];//std::setprecision(0) << a[i*N+j];
                outsfile << "\n";
        }


  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  
  hipStreamDestroy(stream[0]);
  hipStreamDestroy(stream[1]);
  hipStreamDestroy(stream[2]);
  hipStreamDestroy(stream[3]);

  for (k = 0; k < 4; k++) {
    hipFree(d_A[k]);
    hipFree(d_B[k]);
    hipFree(d_C[k]);
  }

}


PluginProxy<GPUStreamVectorAddPlugin> GPUStreamVectorAddPluginProxy = PluginProxy<GPUStreamVectorAddPlugin>("GPUStreamVectorAdd", PluginManager::getInstance());

